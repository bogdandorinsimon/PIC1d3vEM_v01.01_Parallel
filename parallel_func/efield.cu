#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void efieldKernel(
	double* ex, double* ey, double* ez,
	double* by, double* bz, 
	double* jxe, double* jye, double* jze, 
	double* jxi, double* jyi, double* jzi, 
	int m, double c) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= m) {
		return;
	}

	if (index >= 2 && index < m - 3) {
		ex[index] = ex[index] - (jxe[index] + jxi[index]);
		ey[index] = ey[index] - (jye[index] + jyi[index]) - c * (bz[index + 1] - bz[index]);
		ez[index] = ez[index] - (jze[index] + jzi[index]) + c * (by[index + 1] - by[index]);
	}


	if (index == m - 1) {
		ex[index] = ex[4];
		ey[index] = ey[4];
		ez[index] = ez[4];
	}

	if (index == m - 2) {
		ex[index] = ex[3];
		ey[index] = ey[3];
		ez[index] = ez[3];
	}

	if (index == m - 3) {
		ex[index] = ex[2];
		ey[index] = ey[2];
		ez[index] = ez[2];
	}

	switch (index) {
	case 0:
		ex[index] = ex[m - 5];
		ey[index] = ey[m - 5];
		ez[index] = ez[m - 5];
		break;
	case 1:
		ex[index] = ex[m - 4];
		ey[index] = ey[m - 4];
		ez[index] = ez[m - 4];
		break;
	}
}

hipError_t efieldWithCuda(
	double* h_ex, double* h_ey, double* h_ez, 
	double* h_by, double* h_bz, 
	double* h_jxe, double* h_jye, double* h_jze, 
	double* h_jxi, double* h_jyi, double* h_jzi, 
	int m, double c) {

	double* d_ex, * d_ey, * d_ez;
	double* d_by, * d_bz;
	double* d_jxe, * d_jye, * d_jze;
	double* d_jxi, * d_jyi, * d_jzi;

	hipError_t cudaStatus;

	const unsigned long ARRAY_BYTES = m * sizeof(double);
	const unsigned long BLOCK_SIZE = 256;
	const unsigned long NUM_OF_BLOCKS = (m - 1) / BLOCK_SIZE + 1;

	hipMalloc((void**)&d_ex, ARRAY_BYTES);
	hipMalloc((void**)&d_ey, ARRAY_BYTES);
	hipMalloc((void**)&d_ez, ARRAY_BYTES);

	hipMalloc((void**)&d_by, ARRAY_BYTES);
	hipMalloc((void**)&d_bz, ARRAY_BYTES);

	hipMalloc((void**)&d_jxe, ARRAY_BYTES);
	hipMalloc((void**)&d_jye, ARRAY_BYTES);
	hipMalloc((void**)&d_jze, ARRAY_BYTES);

	hipMalloc((void**)&d_jxi, ARRAY_BYTES);
	hipMalloc((void**)&d_jyi, ARRAY_BYTES);
	hipMalloc((void**)&d_jzi, ARRAY_BYTES);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_ex, h_ex, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_ey, h_ey, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_ez, h_ez, ARRAY_BYTES, hipMemcpyHostToDevice);

	hipMemcpy(d_by, h_by, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_bz, h_bz, ARRAY_BYTES, hipMemcpyHostToDevice);

	hipMemcpy(d_jxe, h_jxe, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_jye, h_jye, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_jze, h_jze, ARRAY_BYTES, hipMemcpyHostToDevice);

	hipMemcpy(d_jxi, h_jxi, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_jyi, h_jyi, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_jzi, h_jzi, ARRAY_BYTES, hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMemcpyHostToDevice failed!");
		goto Error;
	}

	efieldKernel << <NUM_OF_BLOCKS, BLOCK_SIZE>> > (d_ex, d_ey, d_ez, d_by, d_bz, d_jxe, d_jye, d_jze, d_jxi, d_jyi, d_jzi, m, c);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: efieldKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching efieldKernel!\n", cudaStatus);
		goto Error;
	}

	hipMemcpy(h_ex, d_ex, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(h_ey, d_ey, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(h_ez, d_ez, ARRAY_BYTES, hipMemcpyDeviceToHost);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(d_ex);
	hipFree(d_ey);
	hipFree(d_ez);
	hipFree(d_by);
	hipFree(d_bz);
	hipFree(d_jxe);
	hipFree(d_jye);
	hipFree(d_jze);
	hipFree(d_jxi);
	hipFree(d_jyi);
	hipFree(d_jzi);

	return cudaStatus;
}
