#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void efieldKernel(
	double* ex, double* ey, double* ez,
	double* by, double* bz, 
	double* jxe, double* jye, double* jze, 
	double* jxi, double* jyi, double* jzi, 
	int m, double c) {
	int i = threadIdx.x;

	if (i >= 2 && i < m - 3) {
		ex[i] = ex[i] - (jxe[i] + jxi[i]);
		ey[i] = ey[i] - (jye[i] + jyi[i]) - c * (bz[i + 1] - bz[i]);
		ez[i] = ez[i] - (jze[i] + jzi[i]) + c * (by[i + 1] - by[i]);
	}


	if (i == m - 1) {
		ex[i] = ex[4];
		ey[i] = ey[4];
		ez[i] = ez[4];
	}

	if (i == m - 2) {
		ex[i] = ex[3];
		ey[i] = ey[3];
		ez[i] = ez[3];
	}

	if (i == m - 3) {
		ex[i] = ex[2];
		ey[i] = ey[2];
		ez[i] = ez[2];
	}

	switch (i) {
	case 0:
		ex[i] = ex[m - 5];
		ey[i] = ey[m - 5];
		ez[i] = ez[m - 5];
		break;
	case 1:
		ex[i] = ex[m - 4];
		ey[i] = ey[m - 4];
		ez[i] = ez[m - 4];
		break;
	}
}

hipError_t efieldWithCuda(
	double* h_ex, double* h_ey, double* h_ez, 
	double* h_by, double* h_bz, 
	double* h_jxe, double* h_jye, double* h_jze, 
	double* h_jxi, double* h_jyi, double* h_jzi, 
	int m, double c) {

	double* d_ex, * d_ey, * d_ez;
	double* d_by, * d_bz;
	double* d_jxe, * d_jye, * d_jze;
	double* d_jxi, * d_jyi, * d_jzi;

	hipError_t cudaStatus;

	const unsigned ARRAY_BITES = m * sizeof(double);
	hipMalloc((void**)&d_ex, ARRAY_BITES);
	hipMalloc((void**)&d_ey, ARRAY_BITES);
	hipMalloc((void**)&d_ez, ARRAY_BITES);

	hipMalloc((void**)&d_by, ARRAY_BITES);
	hipMalloc((void**)&d_bz, ARRAY_BITES);

	hipMalloc((void**)&d_jxe, ARRAY_BITES);
	hipMalloc((void**)&d_jye, ARRAY_BITES);
	hipMalloc((void**)&d_jze, ARRAY_BITES);

	hipMalloc((void**)&d_jxi, ARRAY_BITES);
	hipMalloc((void**)&d_jyi, ARRAY_BITES);
	hipMalloc((void**)&d_jzi, ARRAY_BITES);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_ex, h_ex, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_ey, h_ey, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_ez, h_ez, ARRAY_BITES, hipMemcpyHostToDevice);

	hipMemcpy(d_by, h_by, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_bz, h_bz, ARRAY_BITES, hipMemcpyHostToDevice);

	hipMemcpy(d_jxe, h_jxe, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_jye, h_jye, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_jze, h_jze, ARRAY_BITES, hipMemcpyHostToDevice);

	hipMemcpy(d_jxi, h_jxi, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_jyi, h_jyi, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_jzi, h_jzi, ARRAY_BITES, hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMemcpyHostToDevice failed!");
		goto Error;
	}

	efieldKernel << <1, m >> > (d_ex, d_ey, d_ez, d_by, d_bz, d_jxe, d_jye, d_jze, d_jxi, d_jyi, d_jzi, m, c);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: efieldKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching efieldKernel!\n", cudaStatus);
		goto Error;
	}

	hipMemcpy(h_ex, d_ex, ARRAY_BITES, hipMemcpyDeviceToHost);
	hipMemcpy(h_ey, d_ey, ARRAY_BITES, hipMemcpyDeviceToHost);
	hipMemcpy(h_ez, d_ez, ARRAY_BITES, hipMemcpyDeviceToHost);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "efieldWithCuda: hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(d_ex);
	hipFree(d_ey);
	hipFree(d_ez);
	hipFree(d_by);
	hipFree(d_bz);
	hipFree(d_jxe);
	hipFree(d_jye);
	hipFree(d_jze);
	hipFree(d_jxi);
	hipFree(d_jyi);
	hipFree(d_jzi);

	return cudaStatus;
}
