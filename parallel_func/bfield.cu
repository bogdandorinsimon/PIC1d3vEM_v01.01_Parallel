﻿#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void bfieldKernel(double *by, double *bz, double *ey, double *ez, int m, double c) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= m) {
		return;
	}

	if (index >= 3 && index <= m - 3) {
		by[index] = by[index] + 0.5 * c * (ez[index] - ez[index - 1]);
		bz[index] = bz[index] - 0.5 * c * (ey[index] - ey[index - 1]);
	}

	if (index == m - 1) {
		by[index] = by[4];
		bz[index] = bz[4];
	}

	if (index == m - 2) {
		by[index] = by[3];
		bz[index] = bz[3];
	}

	switch (index) {
	case 0:
		by[index] = by[m - 5];
		bz[index] = bz[m - 5];
		break;
	case 1:
		by[index] = by[m - 4];
		bz[index] = bz[m - 4];
		break;
	case 2:
		by[index] = by[m - 3];
		bz[index] = bz[m - 3];
		break;
	}
}

hipError_t bfieldWithCuda(double *h_by, double *h_bz, double *h_ey, double *h_ez, int m, double c) {
	double *d_by, *d_bz, *d_ey, *d_ez;
	hipError_t cudaStatus;

	const unsigned long ARRAY_BYTES = m * sizeof(double);
	const unsigned long BLOCK_SIZE = 256;
	const unsigned long NUM_OF_BLOCKS = (m - 1) / BLOCK_SIZE + 1;

	hipMalloc((void**) &d_by, ARRAY_BYTES);
	hipMalloc((void**) &d_bz, ARRAY_BYTES);
	hipMalloc((void**) &d_ey, ARRAY_BYTES);
	hipMalloc((void**) &d_ez, ARRAY_BYTES);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_by, h_by, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_bz, h_bz, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_ey, h_ey, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_ez, h_ez, ARRAY_BYTES, hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMemcpyHostToDevice failed!");
		goto Error;
	}

	bfieldKernel<<<NUM_OF_BLOCKS, BLOCK_SIZE>>>(d_by, d_bz, d_ey, d_ez, m, c);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: bfieldKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bfieldKernel!\n", cudaStatus);
		goto Error;
	}

	hipMemcpy(h_by, d_by, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(h_bz, d_bz, ARRAY_BYTES, hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(d_by);
	hipFree(d_bz);
	hipFree(d_ey);
	hipFree(d_ez);

	return cudaStatus;
}
