﻿#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void bfieldKernel(double *by, double *bz, double *ey, double *ez, int m, double c) {
	int i = threadIdx.x;

	if (i >= 3 && i <= m - 3) {
		by[i] = by[i] + 0.5 * c * (ez[i] - ez[i - 1]);
		bz[i] = bz[i] - 0.5 * c * (ey[i] - ey[i - 1]);
	}

	if (i == m - 1) {
		by[i] = by[4];
		bz[i] = bz[4];
	}

	if (i == m - 2) {
		by[i] = by[3];
		bz[i] = bz[3];
	}

	switch (i) {
	case 0:
		by[i] = by[m - 5];
		bz[i] = bz[m - 5];
		break;
	case 1:
		by[i] = by[m - 4];
		bz[i] = bz[m - 4];
		break;
	case 2:
		by[i] = by[m - 3];
		bz[i] = bz[m - 3];
		break;
	}
}

hipError_t bfieldWithCuda(double *h_by, double *h_bz, double *h_ey, double *h_ez, int m, double c) {
	double *d_by, *d_bz, *d_ey, *d_ez;
	hipError_t cudaStatus;

	const unsigned ARRAY_BITES = m * sizeof(double);
	hipMalloc((void**) &d_by, ARRAY_BITES);
	hipMalloc((void**) &d_bz, ARRAY_BITES);
	hipMalloc((void**) &d_ey, ARRAY_BITES);
	hipMalloc((void**) &d_ez, ARRAY_BITES);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_by, h_by, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_bz, h_bz, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_ey, h_ey, ARRAY_BITES, hipMemcpyHostToDevice);
	hipMemcpy(d_ez, h_ez, ARRAY_BITES, hipMemcpyHostToDevice);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMemcpyHostToDevice failed!");
		goto Error;
	}

	bfieldKernel<<<1, m>>>(d_by, d_bz, d_ey, d_ez, m, c);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: bfieldKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bfieldKernel!\n", cudaStatus);
		goto Error;
	}

	hipMemcpy(h_by, d_by, ARRAY_BITES, hipMemcpyDeviceToHost);
	hipMemcpy(h_bz, d_bz, ARRAY_BITES, hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bfieldWithCuda: hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(d_by);
	hipFree(d_bz);
	hipFree(d_ey);
	hipFree(d_ez);

	return cudaStatus;
}
