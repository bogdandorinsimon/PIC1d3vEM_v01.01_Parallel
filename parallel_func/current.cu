#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void currentKernel(double* jxe_s, double* jye_s, double* jze_s, double* jxi_s, double* jyi_s, double* jzi_s, double* x, double* vx, double* vy, \
	double* vz, double qse, double qsi, int np, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Hello world from index %d", index);
}

hipError_t currentWithCuda(double* h_jxe_s, double* h_jye_s, double* h_jze_s, double* h_jxi_s, double* h_jyi_s, double* h_jzi_s, double* h_x, double* h_vx, double* h_vy, \
	double* h_vz, double qse, double qsi, int np, int m) {
	double* d_jxe_s, *d_jye_s, *d_jze_s, *d_jxi_s,* d_jyi_s,* d_jzi_s, *d_x, *d_vx,* d_vy, *d_vz;
	hipError_t cudaStatus;

	const unsigned NUMBER_OF_PARTICLES = 2 * np;
	const unsigned ARRAY_BYTES_CELLS = m * sizeof(double);
	const unsigned ARRAY_BYTES_PARTICLES = NUMBER_OF_PARTICLES * sizeof(double);
	const unsigned BLOCK_SIZE = 256;
	const unsigned NUM_OF_BLOCKS = (NUMBER_OF_PARTICLES - 1) / BLOCK_SIZE;

	hipMalloc((void**)&d_jxe_s, ARRAY_BYTES_CELLS);
	hipMalloc((void**)&d_jye_s, ARRAY_BYTES_CELLS);
	hipMalloc((void**)&d_jze_s, ARRAY_BYTES_CELLS);
	hipMalloc((void**)&d_jxi_s, ARRAY_BYTES_CELLS);
	hipMalloc((void**)&d_jyi_s, ARRAY_BYTES_CELLS);
	hipMalloc((void**)&d_jzi_s, ARRAY_BYTES_CELLS);

	hipMalloc((void**)&d_x, ARRAY_BYTES_PARTICLES);
	hipMalloc((void**)&d_vx, ARRAY_BYTES_PARTICLES);
	hipMalloc((void**)&d_vy, ARRAY_BYTES_PARTICLES);
	hipMalloc((void**)&d_vz, ARRAY_BYTES_PARTICLES);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "currentWithCuda: hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(d_jxe_s, h_jxe_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);
	hipMemcpy(d_jye_s, h_jye_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);
	hipMemcpy(d_jze_s, h_jze_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);
	hipMemcpy(d_jxi_s, h_jxi_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);
	hipMemcpy(d_jyi_s, h_jyi_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);
	hipMemcpy(d_jzi_s, h_jzi_s, ARRAY_BYTES_CELLS, hipMemcpyHostToDevice);

	hipMemcpy(d_x, h_x, ARRAY_BYTES_PARTICLES, hipMemcpyHostToDevice);
	hipMemcpy(d_vx, h_vx, ARRAY_BYTES_PARTICLES, hipMemcpyHostToDevice);
	hipMemcpy(d_vy, h_vy, ARRAY_BYTES_PARTICLES, hipMemcpyHostToDevice);
	hipMemcpy(d_vz, h_vz, ARRAY_BYTES_PARTICLES, hipMemcpyHostToDevice);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "currentWithCuda: hipMemcpyHostToDevice failed!");
		goto Error;
	}

	currentKernel<<<NUM_OF_BLOCKS, BLOCK_SIZE >> > (d_jxe_s, d_jye_s, d_jze_s, d_jxi_s, d_jyi_s, d_jzi_s, d_x, d_vx, d_vy, d_vz, qse, qsi, np, m);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "currentWithCuda: currentKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching currentKernel!\n", cudaStatus);
		goto Error;
	}

	/* hipMemcpy(h_x, d_x, ARRAY_BYTES_PARTICLES, hipMemcpyDeviceToHost);
	hipMemcpy(h_vx, d_vx, ARRAY_BYTES_PARTICLES, hipMemcpyDeviceToHost);
	hipMemcpy(h_vy, d_vy, ARRAY_BYTES_PARTICLES, hipMemcpyDeviceToHost);
	hipMemcpy(h_vz, d_vz, ARRAY_BYTES_PARTICLES, hipMemcpyDeviceToHost);
	*/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "currentWithCuda: hipMemcpyDeviceToHost failed!");
		goto Error;
	}

Error:
	hipFree(d_jxe_s);
	hipFree(d_jye_s);
	hipFree(d_jze_s);
	hipFree(d_jxi_s);
	hipFree(d_jyi_s);
	hipFree(d_jzi_s);

	hipFree(d_x);
	hipFree(d_vx);
	hipFree(d_vy);
	hipFree(d_vz);

	return cudaStatus;
}
